#include "hip/hip_runtime.h"
// Elif Ceylan - TP6 CUDA

#include "stdio.h"
#include <iostream>
#include "Array2D.hpp"
#include <fstream>

__device__ int divergence(float2 z0, float2 c, double bound, int imax){
    float2 z = make_float2(z0.x, z0.y);
    for(int i=0; i<imax; i++){
        float norm = sqrt(z.x*z.x + z.y*z.y);
        if(norm > bound) return i;
        float zx = z.x, zy = z.y;
        z.x = zx*zx - zy*zy + c.x;
        z.y = 2*zx*zy + c.y;
    }
    return imax;
}


__device__ float2 coord2cplx(float2 ll, float2 ur, int x, int y, int2 d){
    float2 res = make_float2( ll.x + x*(ur.x - ll.x)/d.x, -(ll.y + y*(ur.y - ll.y)/d.y ));
    return res;
}


__global__ void julia(float2 ll, float2 ur, float2 c, int imax, int2 d, int* A){
    int2 index = make_int2(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y);
    if (index.x < d.x && index.y < d.y){
        A[index.y*d.x+index.x] = divergence( coord2cplx(ll, ur, index.x, index.y, d), c, 2.0, imax);
    }
}


void writePgm(Array2D<int>& d, int imax, std::string filename){
    std::ofstream file;
    file.open (filename);
    file << "P2" << std::endl;
    file << d.sizeX() << " " << d.sizeY() << std::endl;
    file << imax << std::endl;
    for(int y=0; y<d.sizeY(); y++){
        for(int x=0; x<d.sizeX(); x++){
            file << d(x, y) << " ";
        }
        file << std::endl;
    }
    file.close();
}

// Main function to be executed on host
int main(int argc, char** argv)
{
    float2 lowerLeft = make_float2(std::stof(argv[1]), std::stof(argv[2]));
    float2 upperRight = make_float2(std::stof(argv[3]), std::stof(argv[4]));
    float2 c = make_float2(std::stof(argv[5]), std::stof(argv[6]));
    int imax = std::stoi(argv[7]);
    int2 domain = make_int2(std::stoi(argv[8]), std::stoi(argv[9]));
    std::string filename(argv[10]);

    size_t domainSize = domain.x*domain.y*sizeof(int);

    // host memory allocation
    Array2D<int> hostDomain(domain.x, domain.y, 0);

    // device memory allocation
    int* deviceDomain;
    hipMalloc(&deviceDomain, domainSize);

    // host
    hipMemset(deviceDomain, 0, domainSize);

    // preparation de la configuration d'execution
    dim3 dimBlock(16, 16);
    dim3 dimGrid((domain.x + dimBlock.x - 1) / dimBlock.x, (domain.y + dimBlock.y - 1) / dimBlock.y);
    //std::cout << "dimGrid : " << dimGrid.x << ", " << dimGrid.y << std::endl;

    std::cout << "gpu: " << dimGrid.x << ", " << dimGrid.y << "; " << domain.x << "; (" << c.x << "," << c.y <<"); " << imax << std::endl;

    // variables necéssaires à la mesure du temps
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // execution du kernel + mesure de celle-ci
    hipEventRecord(start, 0);
    julia<<<dimGrid, dimBlock>>>(lowerLeft, upperRight, c, imax, domain, deviceDomain);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);


    std::cout << "temps de calcul: " << time << std::endl;

    // variables necéssaires à la mesure du temps
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // copie des donnees du device vers le host
    hipMemcpy(hostDomain.data(), deviceDomain, domainSize, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);


    std::cout << "temps de copie des données: " << time << std::endl;

    writePgm(hostDomain, imax, filename);

    // liberation de la memoire du device
    hipFree(deviceDomain);
}
