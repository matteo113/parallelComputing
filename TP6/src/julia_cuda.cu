#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include "Array2D.hpp"
#include <fstream>
#include <hip/hip_runtime.h>
#include <math.h>


// commpute the ammount of iteration before reaching divergence
__device__ int divergence(float2 z0, float2 c, double bound, int imax){
    float2 z = make_float2(z0.x, z0.y);
    for(int i=0; i<imax; i++){
        float norm = sqrt(z.x*z.x + z.y*z.y);
        if(norm > bound) return i;
        float zx = z.x, zy = z.y;
        z.x = zx*zx - zy*zy + c.x;
        z.y = 2*zx*zy + c.y;
    }
    return imax;
}

// convert discrete dcoordinate to the complex plan
__device__ float2 coord2cplx(float2 ll, float2 ur, int x, int y, int2 d){
    float2 res = make_float2( ll.x + x*(ur.x - ll.x)/d.x, -(ll.y + y*(ur.y - ll.y)/d.y ));
    return res;
}


// julia computation inspired of "Addition d'une constante sur une matrice" on 
// http://unige.ch/spc/en/info/teaching/parallelisme/programmation-sur-gpu-avec-cuda/
__global__ void julia(float2 ll, float2 ur, float2 c, int imax, int2 d, int* deviceDomain){
    int2 index = make_int2(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y);
    if (index.x < d.x && index.y < d.y){
        deviceDomain[index.y*d.x+index.x] = divergence( coord2cplx(ll, ur, index.x, index.y, d), c, 2.0, imax);
    }
}

// an Array2D as a PGM image
void writePgm(Array2D<int>& d, int imax, std::string filename){
    std::ofstream file;
    file.open (filename);
    file << "P2" << std::endl;
    file << d.sizeX() << " " << d.sizeY() << std::endl;
    file << imax << std::endl;
    for(int y=0; y<d.sizeY(); y++){
        for(int x=0; x<d.sizeX(); x++){
            file << d(x, y) << " ";
        }
        file << std::endl;
    }
    file.close();
}

// Main function to be executed on host
int main(int argc, char** argv)
{
    float2 lowerLeft = make_float2(std::stof(argv[1]), std::stof(argv[2]));
    float2 upperRight = make_float2(std::stof(argv[3]), std::stof(argv[4]));
    float2 c = make_float2(std::stof(argv[5]), std::stof(argv[6]));
    int imax = std::stoi(argv[7]);
    int2 domain = make_int2(std::stoi(argv[8]), std::stoi(argv[9]));
    std::string filename(argv[10]);

    size_t domainSize = domain.x*domain.y*sizeof(int);

    // host memory allocation and setting
    Array2D<int> hostDomain(domain.x, domain.y, 0);

    // device memory allocation
    int* deviceDomain;
    hipMalloc(&deviceDomain, domainSize);

    // device memory setting
    hipMemset(deviceDomain, 0, domainSize);

    // exec configuration from http://unige.ch/spc/en/info/teaching/parallelisme/programmation-sur-gpu-avec-cuda/
    dim3 dimBlock(16, 16);
    dim3 dimGrid((domain.x + dimBlock.x - 1) / dimBlock.x, (domain.y + dimBlock.y - 1) / dimBlock.y);

    // setting of the time measurement 
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // kernel execution
    hipEventRecord(start, 0);
    julia<<<dimGrid, dimBlock>>>(lowerLeft, upperRight, c, imax, domain, deviceDomain);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);


    std::cout << "compute : " << time << std::endl;

    // reset of the time measurment
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Device to host data copy after computation
    hipMemcpy(hostDomain.data(), deviceDomain, domainSize, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);


    std::cout << "data copy: " << time << std::endl;

    writePgm(hostDomain, imax, filename);

    // device memory free
    hipFree(deviceDomain);
}
